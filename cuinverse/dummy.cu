#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>

#include <hip/hip_runtime_api.h>

#include <hipblas.h>

#define BLOCK_SIZE 16

__global__ void kernel()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    printf("Hello from block %d, thread %d\n", i, j);
}

__device__ void kernelcall()
{
    kernel<<<10, 1>>>();
}

__global__ void mainkernel()
{
    kernelcall();
}

int main()
{
    mainkernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}